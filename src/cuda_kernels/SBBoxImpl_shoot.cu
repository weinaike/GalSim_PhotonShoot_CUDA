#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "SBBoxImpl_shoot.h"
#include "galsim/PhotonArray.h"

#include "SBBoxImpl_shoot.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#ifdef ENABLE_CUDA

#include "cuda_check.h"

struct SBounds {
    double width, height;
};

template <typename T>
__global__ void SBBoxImpl_shoot_kernel(
    T* x, T* y, T* flux, const SBounds bounds, const double fluxPerPhoton, int N, hiprandState* state)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        hiprandState localState = state[id];
        double randX = hiprand_uniform(&localState) - 0.5;
        double randY = hiprand_uniform(&localState) - 0.5;
        x[id] = randX * bounds.width;
        y[id] = randY * bounds.height;
        flux[id] = fluxPerPhoton;
        state[id] = localState;
    }
}

// CUDA kernel for setting up the random states
__global__ void setup_kernel(hiprandState *state, int seed, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
        hiprand_init(seed, id, 0, &state[id]);
}


namespace galsim {
        
    void SBBoxImpl_shoot_cuda(PhotonArray& photons, double width, double height, double flux, UniformDeviate ud)
    {
        const int N = photons.size();
        double fluxPerPhoton = flux / N;

        // 在GPU上分配内存
        double* d_x = photons.getXArrayGpu();
        double* d_y = photons.getYArrayGpu();
        double* d_flux = photons.getFluxArrayGpu();
        hiprandState* d_state;

        hipMalloc((void**)&d_state, N * sizeof(hiprandState));

        // 初始化hiprand状态
        int blockSize = 256;
        int gridSize = (N + blockSize - 1) / blockSize;
        setup_kernel<<<gridSize, blockSize>>>(d_state, time(0), N);

        // 调用CUDA核函数
        SBounds bounds = {width, height};
        SBBoxImpl_shoot_kernel<<<gridSize, blockSize>>>(d_x, d_y, d_flux, bounds, fluxPerPhoton, N, d_state);
        CUDA_CHECK_RETURN(hipDeviceSynchronize());

        hipFree(d_state);
    }

}

#endif